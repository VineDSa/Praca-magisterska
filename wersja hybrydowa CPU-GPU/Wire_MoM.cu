#include "hip/hip_runtime.h"
#include "Wire_MoM.h"

//using namespace std;

double cspeed = 2.99792448E+8;;
double pi = std::acos(-1.);

struct Timer
{
	chrono::time_point<chrono::high_resolution_clock> start, end;
	chrono::duration<float> duration;
	char* info = "None";
	int rank = 9999;
	Timer()
	{
		start = chrono::high_resolution_clock::now();
	}

	~Timer()
	{
		end = chrono::high_resolution_clock::now();
		duration = end - start;
		float s = duration.count();
		std::cout << std::endl << " Process - " << rank << " " << info << " took " << std::setprecision(4) << s << "s" << std::endl;
	}
};

int main(int argc, char** argv)
{
	Timer timer;
	int rank;
	int num_procs;

	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &num_procs);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	timer.rank = rank;
	timer.info = "Whole calculations";

	if (rank == BROADCASTER_RANK)
	{
		std::cout << std::endl;
		std::cout << " -------------------------" << std::endl;
		std::cout << " Implementacja MoM w C++" << std::endl;
		std::cout << " Wersja dla drucikow z PEC" << std::endl;
		std::cout << " Analiza instalacji odgromowych" << std::endl;
		std::cout << " -------------------------" << std::endl;
		std::cout << std::endl;
	}

	//inicjalizacja tablic
	const int nnmax = 3000, ntmax = 5000;
	const int nwmax = 10000, npwmax = 20000, nsmax = nwmax + npwmax;
	int nnod = 0, nw = 0, nt = 0, npnt = 0;
	float *xnod = 0, *ynod = 0, *znod = 0; //wrad;
	int *iwn, *nsw, *iwt;
	float *wrad;
	int nseg;

	//zmienne
	bool wyb;
	int ienv, iexct, iload, ngap, ispc;
	float ampl, phas;


	xnod = new float[nnmax];
	ynod = new float[nnmax];
	znod = new float[nnmax];
	iwn = new int[2 * nwmax];
	wrad = new float[nwmax];
	nsw = new int[nwmax];
	iwt = new int[3 * ntmax];

	//zerowanie tablic
	if (rank == BROADCASTER_RANK)
	{
		for (int i = 0; i < nnmax; i++)
		{
			xnod[i] = 0.;
			ynod[i] = 0.;
			znod[i] = 0.;
		}
		for (int i = 0; i < nwmax; i++)
		{
			iwn[i] = 0;
			iwn[i + nwmax] = 0;
			wrad[i] = 0.;
			nsw[i] = 0;
		}
		for (int i = 0; i < ntmax; i++)
		{
			iwt[i] = 0;
			iwt[i + ntmax] = 0;
			iwt[i + 2 * ntmax] = 0;
		}
	}
	// wczytanie pliku geo
	if(rank == BROADCASTER_RANK)
		geoinp(nnmax, nwmax, ntmax, nnod, xnod, ynod, znod, nw, iwn, wrad, nsw, nt, iwt);

	wyb = true;
	// Wersja umo�liwiaj�ca uwzgl�dnienie PEC dla z=0 (ienv=0)
	//ienv = 2;
	ienv = 1;
	if (rank == BROADCASTER_RANK)
	{
		while (wyb)
		{
			std::cout << " Environment: free space       --> 1" << std::endl;
			std::cout << "              pec ground (z=0) --> 2" << std::endl;
			std::cout << " Select indicator (1 or 2):  " << ienv << std::endl;
			//cin >> ienv;
			if (ienv == 1 || ienv == 2) wyb = false;
		}
	}
	int isym;
	float sgnenv, sgnx, sgny, sgnz;
	if (ienv == 1)
	{
		sgnenv = 1.;
		isym = 0;
		sgnx = 1.;
		sgny = 1.;
		sgnz = 1.;
	}
	else
	{
		sgnenv = -1.;
		isym = 3;
		sgnx = 1.;
		sgny = 1.;
		sgnz = -1.;
	}

	//w�z�y
	float *x, *y, *z;
	// wi�ksza liczba punkt�w ni� nnod poniewa� w procedurze config tworzone s� nadmiarowe w�z�y, kt�re potem s� eliminowane
	x = new float[nsmax];
	y = new float[nsmax];
	z = new float[nsmax];

	//tablice zwi�zane z drucikami
	int *ise, *isp, *ips, *ipc, *iwe;
	float *rad;
	ise = new int[2 * nsmax];
	isp = new int[2 * nsmax];
	ips = new int[2 * nsmax];
	ipc = new int[nsmax];
	iwe = new int[2 * nsmax];
	rad = new float[nsmax];

	int npls = 0, nplsw = 0;

	if (rank == BROADCASTER_RANK)
	{
		//zerowanie tablic
		for (int i = 0; i < nsmax; i++)
		{
			x[i] = 0.;
			y[i] = 0.;
			z[i] = 0.;
			ise[i] = 0;
			ise[i + nsmax] = 0;
			isp[i] = 0;
			isp[i + nsmax] = 0;
			iwe[i] = 0;
			iwe[i + nsmax] = 0;
			rad[i] = 0.;
			ips[i] = 0;
			ips[i + nsmax] = 0;
			ipc[i] = 0;
		}
	}

	// konfiguracja modelu - wype�nienie wszystkich tablic zwi�zanych z przewodami
	if(rank == BROADCASTER_RANK)
		config(npnt, nnod, xnod, ynod, znod, x, y, z, nw, iwn, wrad, nsw, ise, isp, ips, ipc, iwe, rad, npls, nplsw, nseg, nsmax, nwmax, ienv);

	// Pobudzenie -> W tej chwili domy�lnie ustawione na 1 -> generator
	wyb = true;
	iexct = 1;
	if(rank == BROADCASTER_RANK)
	{
		while (wyb)
		{
			std::cout << " -------------------" << std::endl;
			std::cout << " EXCITATION: " << std::endl;
			std::cout << " 1 - delta gap source" << std::endl;
			std::cout << " 2 - incident wave" << std::endl;
			std::cout << " Select 1 or 2: " << iexct << std::endl;
			//cin >> iexct;
			//std::cout << std::endl;
			if (iexct == 1 || iexct == 2) wyb = false;
		}
	}
	//if (iexct == 2) std::cout << " ERROR - ten typ pobudzenia nie zaimplementowany!" << std::endl;

	//definiowanie pobudzenia - uproszczone - zak�adamy tylko jeden generator na przewodach -> do analizy LPS
	if (rank == BROADCASTER_RANK)
	{
		if (iexct == 1) // pobudzenie w postaci generatora nexct=1, nexct - liczba generator�w
		{
			wyb = true;
			while (wyb)
			{
				//ngap = 592;
				ngap = 100;
				std::cout << " ------------------------------" << std::endl;
				std::cout << " Source (source zone on wire): " << ngap << std::endl;
				//cin >> ngap; // numer funkcji bazowej na przewodach definiuj�cy miejsce generatora
				if (ngap > 0 && ngap <= npls)
				{
					wyb = false;
				}
				else
				{
					std::cout << "    ERROR (wrong number of source zone)" << std::endl;
				}
			}
			ampl = 1.0;
			std::cout << "       Specify source attributes" << std::endl;
			std::cout << "                 magnitude [V]: " << ampl << std::endl;
			//cin >> ampl;
			ampl = abs(ampl);
			phas = 0.0;
			std::cout << "               phase [degrees]: " << phas << std::endl;
			//cin >> phas;				
		}
	}
	// Obci��enia
	wyb = true;
	//iload = 1;
	iload = 2;
	if (rank == BROADCASTER_RANK)
	{
		while (wyb)
		{
			std::cout << " -----------------------------------" << std::endl;
			std::cout << " Lumped loadings (1-> yes, 2-> no): " << iload << std::endl;
			//cin >> iload;
			if (iload == 1 || iload == 2)
			{
				wyb = false;
			}
			else
			{
				std::cout << "     ERROR!" << std::endl;
			}
		}
	}
	//skalowanie widma -> W tej chwili domy�lnie ustawione na NIE -> 2
	ispc = 2;
	wyb = true;
	if (rank == BROADCASTER_RANK)
	{
		while (wyb)
		{
			std::cout << " -----------------------------------" << std::endl;
			std::cout << " spectrum weight (1-> yes, 2-> no): " << ispc << std::endl;
			//cin >> ispc;
			if (ispc == 1 || ispc == 2)
			{
				wyb = false;
			}
			else
			{
				std::cout << "    ERROR!" << std::endl;
			}
		}
	}
	//wczytanie cz�stotliwo�ci z pliku
	float *freqtab;
	int nfreq;
	if (rank == BROADCASTER_RANK)
	{
		// plik z cz�stotliwo�ciami dla kt�rych wykonywane obliczenia -> pierwszy wiersz zawiera liczb� punkt�w
		fstream freqfile;
		freqfile.open("freq.txt", ios::in);
		freqfile >> nfreq;
		freqtab = new float[nfreq];

		std::cout << std::endl << std::endl;
		std::cout << "  -------------------------------------" << std::endl;
		std::cout << "  Number of frequency points: " << nfreq << std::endl;
		std::cout << "  -------------------------------------" << std::endl;

		for (int i = 0; i < nfreq; i++)
		{
			freqfile >> freqtab[i];
		}

		//resfile.close();
		freqfile.close();
	}
	MPI_Bcast(&npls, 1, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(&nfreq, 1, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);

	if (rank != BROADCASTER_RANK) //only BROADCASTER_RANK allocated memory for freqtab
		freqtab = new float[nfreq];

	MPI_Bcast(freqtab, nfreq, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(x, nsmax, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(y, nsmax, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(z, nsmax, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(rad, nsmax, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(ipc, nsmax, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(ips, 2 * nsmax, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast(ise, 2 * nsmax, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&ienv, 1, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&sgnx, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&sgny, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&sgnz, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&sgnenv, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&iload, 1, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&phas, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&ngap, 1, MPI_INT, BROADCASTER_RANK, MPI_COMM_WORLD);
	MPI_Bcast((void*)&ampl, 1, MPI_FLOAT, BROADCASTER_RANK, MPI_COMM_WORLD);

	//alokacja macierzy z
	clDoubleComplex  *cz;
	cz = new clDoubleComplex[npls*npls];

	// Crete Z matrix in mkl type
	MKL_Complex16* cz_mkl;
	cz_mkl = new MKL_Complex16[npls * npls];

	// alokacja wektora ci_mkl
	MKL_Complex16* ci_mkl;
	ci_mkl = new MKL_Complex16[npls];
	
	double arg;
	int* ipv;
	ipv = new int[npls];

	clDoubleComplex* ci;
	ci = new clDoubleComplex[npls];

	clDoubleComplex* cu;
	cu = new clDoubleComplex[npls];
	for (int i = 0; i < npls; i++)
	{
		cu[i] = Z_ZERO;
	}
	arg = pi * phas / 180.;
	//cu(ngap)=dble(ampl)*cdexp(cj*dtr*phas)
	cu[ngap - 1] = Z_MUL(Z_MAKE(ampl, 0.), Z_MAKE(cos(arg), sin(arg)));
	// zaczynamy p�tl� po cz�stotliwo�ci
	
	for (int i = rank; i < nfreq; i = i+num_procs)
	{
		CalculateByFreq
		(
			i, npls, nfreq, freqtab[i], x, y, z, rad, ipc, ips, ise, nsmax, ienv, 
			sgnx, sgny, sgnz,sgnenv, iload, phas, ngap, ampl, cz, cz_mkl, ci_mkl, ipv, cu, ci, rank
		);
		
	}

	delete[] xnod, ynod, znod;
	delete[] iwn, wrad, nsw, iwt;
	delete[] x, y, z;
	delete[] ise, isp, ips, ipc, iwe, rad;
	delete[] cz;
	delete[] freqtab;

	delete[] cz_mkl, ci_mkl;

	delete[] ipv;
	delete[] ci, cu;
	MPI_Finalize();
	return (0);

}
//---------------------------------------------------------------------------------------------------------------------------
void CalculateByFreq(int i, int npls, int nfreq, float freq,  
	float *x, float* y, float *z, float* rad, int* ipc, int* ips, int* ise,  const int nsmax, 
	int ienv, float sgnx, float sgny, float sgnz, float sgnenv, int iload, 
	float phas, int ngap, float ampl, hipDoubleComplex* cz, MKL_Complex16* cz_mkl, MKL_Complex16* ci_mkl, 
	int* ipv, clDoubleComplex* cu, clDoubleComplex* ci,  int rank)
{
	double ambda, ka;
	clDoubleComplex cload;

	ambda = cspeed / freq * 1E-6;
	ka = (pi + pi) / ambda;

	// zmatrix
	zmatrix_gpu(cz, x, y, z, rad, ipc, ips, ise, nsmax, ka, npls, ienv, sgnx, sgny, sgnz, sgnenv, rank);

	// addloads
	// w tej chwili bez osobnej procedury
	if (iload == 1)
	{
		// dodanie obiążenia do kanału
		cload = Z_MAKE(1.e0, 2. * pi * freq * 4.5);		// 4.5 uH ale freq w MHz	!e - 6
		int nl;
		for (nl = 592 - 1; nl <= 2581 - 1; nl++)  //anis1m_loop.geo
		{
			cz[nl + nl * npls] = Z_ADD(cz[nl + nl * npls], cload);
		}
		//dodanie obciążenia do pętli
		nl = 2613 - 1;	//anis1m_loop.geo
		cload = Z_MAKE(30., 0.);
		cz[nl + nl * npls] = Z_ADD(cz[nl + nl * npls], cload);
		//dodanie obciążenia do krótkiego dipola w środku pętli -> do wyznaczania pola E
		nl = 2582 - 1;	//anis1m_loop.geo
		cload = Z_MAKE(1.e12, 0.);
		cz[nl + nl * npls] = Z_ADD(cz[nl + nl * npls], cload);
	}

	{
		Timer TimerMKL;
		TimerMKL.rank = 0;
		TimerMKL.info = "LAPACKEzgetrs";

		// Conversion Z matrix to lapack_complex_type
		ZmatrixToMKL(cz_mkl, cz, npls);
		// Faktoryzation of impedance matrix		
		LAPACKE_zgetrf(LAPACK_COL_MAJOR, npls, npls, cz_mkl, npls, ipv);

		// Substitution and solution equation
		for (int i = 0; i < npls; i++) ci[i] = cu[i];

		VectorToMKL(ci_mkl, ci, npls);
		LAPACKE_zgetrs(LAPACK_COL_MAJOR, 'n', npls, 1, cz_mkl, npls, ipv, ci_mkl, npls);

		VectorToCom(ci_mkl, ci, npls);
	}
	// przemnożyć prąd przez widmo! - w tej chwili nie jest potrzebne -> przy wyznaczaniu próbek do analizy czasowej

	//ipedancja
	clDoubleComplex csample = Z_DIV(cu[ngap - 1], ci[ngap - 1]);
	std::cout << std::endl << " Process - " << rank << " Freq = " << freq << "\t" << Z_REAL(csample) << "\t" << Z_IMAG(csample) << std::endl;

}

void ZmatrixToMKL(MKL_Complex16* z, clDoubleComplex* cz, int npls)
{
	//int k = 0;
	for (int i = 0; i < npls; i++)
		for (int j = 0; j < npls; j++)
		{
			z[j + i * npls].real = Z_REAL(cz[j + i * npls]);
			z[j + i * npls].imag = Z_IMAG(cz[j + i * npls]);
			//k++;
		}
}

//---------------------------------------------------------------------------------------------------------------------------

void VectorToMKL(MKL_Complex16* cci, clDoubleComplex* ci, int npls)
{
	for (int i = 0; i < npls; i++)
	{
		cci[i].real = Z_REAL(ci[i]);
		cci[i].imag = Z_IMAG(ci[i]);
	}
}

//---------------------------------------------------------------------------------------------------------------------------

void ZmatrixToCom(MKL_Complex16* z, clDoubleComplex* cz, int npls)
{
	//int k = 0;
	for (int i = 0; i < npls; i++)
		for (int j = 0; j < npls; j++)
		{
			cz[j + i * npls] = Z_MAKE(z[j + i * npls].real, z[j + i * npls].imag);
			//k++;
		}
}

//---------------------------------------------------------------------------------------------------------------------------

void VectorToCom(MKL_Complex16* cci, clDoubleComplex* ci, int npls)
{
	for (int i = 0; i < npls; i++) ci[i] = Z_MAKE(cci[i].real, cci[i].imag);
}

//----------------------------------------------------------------------

__device__ double delick_gpu(double bet)
{
	double a0 = (double)1.38629436112e0;
	double a1 = (double)0.09666344259e0;
	double a2 = (double)0.03590092383e0;
	double a3 = (double)0.03742563713e0;
	double a4 = (double)0.01451196212e0;
	double b0 = (double)0.5e0;
	double b1 = (double)0.12498593597e0;
	double b2 = (double)0.06880248576e0;
	double b3 = (double)0.03328355346e0;
	double b4 = (double)0.00441787012e0;
	double res, a, b;
	double am1 = (double)1.0e0 - bet*bet;
	double am12, am13, am14;

	a = a0 + a1*am1;
	b = b0 + b1*am1;

	if (am1 >= (double)1.0e-18){
		am12 = am1*am1;
		a = a + a2*am12;
		b = b + b2*am12;
		if (am1 >= (double)1.0e-12){
			am13 = am12*am1;
			a = a + a3*am13;
			b = b + b3*am13;
			if (am1 >= (double)1.0e-9){
				am14 = am13*am1;
				a = a + a4*am14;
				b = b + b4*am14;
			}
		}
	}

	res = a - b*log(am1);
	return res;

}

//------------------------------------------------

__device__ bool war_gpu_pec_re(int ise_ms1, int ise_ms2, int ise_ns1, int ise_ns2)
{
	if ((ise_ns1 == ise_ms1  && ise_ns2 == ise_ms2) || (ise_ns1 == ise_ms2  && ise_ns2 == ise_ms1))
	{
		return true;
	}
	else {
		return false;
	}
}
//------------------------------------------------

__device__ bool war_gpu_pec_im(int ise_ms1, int ise_ms2, int ise_ns1, int ise_ns2, float z_ise_ms1, float z_ise_ms2)
{
	if (((ise_ns1 == ise_ms1 && ise_ns2 == ise_ms2) && (z_ise_ms1 == 0.0f && z_ise_ms2 == 0.0f))
		|| ((ise_ns1 == ise_ms2 && ise_ns2 == ise_ms1) && (z_ise_ms1 == 0.0f && z_ise_ms2 == 0.0f)))
	{
		return true;
	}
	else {
		return false;
	}
}
//----------------------------------------------------------------------
__global__ void wim_gpu_cknn_cwire1_re1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval1 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval1 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval1 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval1 = Z_ADD(cv1v, cvval1);
			csval1 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval1 = Z_ADD(cv1s, csval1);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval1 = dcmplx(0.0d0,0.0d0)
			//csval1 = dcmplx(0.0d0,0.0d0)
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl1 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
		double tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval1 = csval1/dl
		csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));

		//cvval = dcmplx(tvecl1,0.d0)*cvval1
		hipDoubleComplex cvval = Z_MUL(cvval1, Z_MAKE(tvecl1, (double)0.0e0));
		//csval = csval1
		hipDoubleComplex csval = csval1;

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);


		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = cz_ij;
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}

}
//----------------------------------------------------------------------
__global__ void wim_gpu_cknn_cwire1_re2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		//int ms = ips[i];
		//int mlr = ise[ms-1];
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval1 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval1 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval1 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval1 = Z_ADD(cv1v, cvval1);
			csval1 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval1 = Z_ADD(cv1s, csval1);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval1 = dcmplx(0.0d0,0.0d0)
			//csval1 = dcmplx(0.0d0,0.0d0)
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl1 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
		double tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval1 = csval1/dl
		csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));

		//cvval = dcmplx(tvecl1,0.d0)*cvval1
		hipDoubleComplex cvval = Z_MUL(cvval1, Z_MAKE(tvecl1, (double)0.0e0));
		//csval = csval1
		hipDoubleComplex csval = csval1;

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);
		//hipDoubleComplex cz_ij1 = cz[ i+j*n ];

		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(lmc.x,lmc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] = Z_ADD( cz[ i+j*n ],Z_MAKE((double)10.0f,(double)0.0f) );
		//cz[ i+j*n ] = Z_ADD(cz_ij1, cz_ij );
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}
//----------------------------------------------------------------------
__global__ void wim_gpu_cknn_cwire1_im1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval1 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval1 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval1 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval1 = Z_ADD(cv1v, cvval1);
			csval1 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval1 = Z_ADD(cv1s, csval1);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval1 = dcmplx(0.0d0,0.0d0)
			//csval1 = dcmplx(0.0d0,0.0d0)
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl1 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
		double tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval1 = csval1/dl
		csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));

		//cvval = sgnenv*dcmplx(tvecl1,0.d0)*cvval1
		hipDoubleComplex cvval = Z_MUL(cvval1, Z_MAKE((double)sgnenv*tvecl1, (double)0.0e0));
		//csval = csval1*sgnenv
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), csval1);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);


		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_cknn_cwire1_im2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		//int ms = ips[i];
		//int mlr = ise[ms-1];
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval1 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval1 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval1 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval1 = Z_ADD(cv1v, cvval1);
			csval1 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval1 = Z_ADD(cv1s, csval1);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval1 = dcmplx(0.0d0,0.0d0)
			//csval1 = dcmplx(0.0d0,0.0d0)
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl1 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
		double tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval1 = csval1/dl
		csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));

		//cvval = sgnenv*dcmplx(tvecl1,0.d0)*cvval1
		hipDoubleComplex cvval = Z_MUL(cvval1, Z_MAKE((double)sgnenv*tvecl1, (double)0.0e0));
		//csval = csval1*sgnenv
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), csval1);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);


		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(lmc.x,lmc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] = Z_ADD( cz[ i+j*n ],Z_MAKE((double)10.0f,(double)0.0f) );
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_cknn_cwire2_re1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval2, csval2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j + nsmax];
		int nlr = ise[ns - 1 + nsmax];
		double an = rad[ns - 1];
		double3 rnr = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rn = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		double3 rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			//cvval2 = cvv1a;
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval2 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval2 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval2 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval2 = Z_ADD(cv1v, cvval2);
			csval2 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval2 = Z_ADD(cv1s, csval2);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval2 = dcmplx(0.0d0,0.0d0)
			//csval2 = dcmplx(0.0d0,0.0d0)
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2 
		//!cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2
		double tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval2 = csval2/dl
		csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));

		//cvval = dcmplx(tvecl2,0.d0)*cvval2
		hipDoubleComplex cvval = Z_MUL(cvval2, Z_MAKE(tvecl2, (double)0.0e0));
		//csval = -csval2
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)-1.0e0, (double)0.0e0), csval2);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);


		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[i + (j + kk)*n] = cvval2;
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_cknn_cwire2_re2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval2, csval2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		//int ms = ips[i];
		//int mlr = ise[ms-1];
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j + nsmax];
		int nlr = ise[ns - 1 + nsmax];
		double an = rad[ns - 1];
		double3 rnr = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rn = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		double3 rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval2 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval2 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval2 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval2 = Z_ADD(cv1v, cvval2);
			csval2 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval2 = Z_ADD(cv1s, csval2);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval2 = dcmplx(0.0d0,0.0d0)
			//csval2 = dcmplx(0.0d0,0.0d0)
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2 
		//!cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2
		double tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval2 = csval2/dl
		csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));

		//cvval = dcmplx(tvecl2,0.d0)*cvval2
		hipDoubleComplex cvval = Z_MUL(cvval2, Z_MAKE(tvecl2, (double)0.0e0));
		//csval = -csval2
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)-1.0e0, (double)0.0e0), csval2);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);
		//hipDoubleComplex cz_ij1 = cz[ i+j*n ];

		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.z,lmc.z);
		//cz[ i+j*n ] = cvval;
		//cz[ i+j*n ] = Z_MAKE(tvecl2,(double)0.0e0);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] = cz_ij;
		//cz[ i+j*n ] = Z_ADD( cz[ i+j*n ],Z_MAKE((double)10.0f,(double)0.0f) );
		//cz[ i+j*n ] = Z_ADD(cz_ij1, cz_ij );
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_cknn_cwire2_im1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval2, csval2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j + nsmax];
		int nlr = ise[ns - 1 + nsmax];
		double an = rad[ns - 1];
		double3 rnr = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rn = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		double3 rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval2 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval2 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval2 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval2 = Z_ADD(cv1v, cvval2);
			csval2 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval2 = Z_ADD(cv1s, csval2);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval2 = dcmplx(0.0d0,0.0d0)
			//csval2 = dcmplx(0.0d0,0.0d0)
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2 
		//!cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2
		double tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval2 = csval2/dl
		csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));

		//cvval = sgnenv*dcmplx(tvecl2,0.d0)*cvval2
		hipDoubleComplex cvval = Z_MUL(cvval2, Z_MAKE((double)sgnenv*tvecl2, (double)0.0e0));
		//csval = -sgnenv*csval2
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)-1.0e0*(double)sgnenv, (double)0.0e0), csval2);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);


		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_cknn_cwire2_im2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval2, csval2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		//int ms = ips[i];
		//int mlr = ise[ms-1];
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j + nsmax];
		int nlr = ise[ns - 1 + nsmax];
		double an = rad[ns - 1];
		double3 rnr = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rn = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		double3 rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4
			//  res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1
			//  res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3		    
			// --------------- cv1 ------------------
			//dcqg_a = 0.5d0*(0.0d0+dlh)
			//dcqg_b = dlh-0.0d0
			double dcqg_a = (double)0.5e0*dlh;
			double dcqg_b = dlh;
			//
			double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			double cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			double cknn_r = (double)0.5e0*dl - cknn_s;
			double cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			hipDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			double cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			double cknn_res = delick_gpu(cknn_bet);
			double cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			hipDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			hipDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			hipDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv1v = dcqg_b*( cvv1a+cvv1b )
			//cv1s = dcqg_b*( csv1a+csv1b ) 
			hipDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv1 ------------------


			//call dcqg(dlh,dl,cknn,cv2,4)	      
			//nord=4
			// --------------- cv2 ------------------
			dcqg_a = (double)0.5e0*(dlh + dl);
			dcqg_b = dl - dlh;
			//
			dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
			//y=.17392742256872693d0*(f(a+c)+f(a-c))
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			///cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			///cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = cv;
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)          ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			csv1a = cv;
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//!cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))        !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)            ! pot. scalarny 
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1a = .17392742256872693d0*( cv + csv1a )
			csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
			// ---- cknn ---- 

			dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
			//y=b*(y+.32607257743127307d0*(f(a+c)+f(a-c))) 
			//s=a+c
			cknn_s = dcqg_a + dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))         !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//cv = Z_MUL( Z_MAKE(cknn_s/dl,(double)0.0e0),cv );
			cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)               ! pot. scalarny 
			//cv = Z_ADD( cknn_cbnd1,Z_MAKE(cknn_bnd2,(double)0.0e0) );
			csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			// ---- cknn ----
			//s=a-c
			cknn_s = dcqg_a - dcqg_c;
			// ---- cknn ----
			cknn_r = (double)0.5e0*dl - cknn_s;
			cknn_rr = sqrt(an*an + cknn_r*cknn_r);
			//cknn_cbnd1 = ( dcos(ka*cknn_rr) - cj*dsin(ka*cknn_rr) - cone )/cknn_rr
			//cknn_cbnd1 = dcmplx( cos(ka*cknn_rr)-1.0d0, - sin(ka*cknn_rr) )/cknn_rr
			cknn_cbnd1 = Z_MAKE((cos(ka*cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka*cknn_rr) / cknn_rr);
			cknn_bet = (an + an) / sqrt((double)4.0e0*an*an + cknn_r*cknn_r);
			//call delick_gpu(cknn_bet,cknn_res)
			cknn_res = delick_gpu(cknn_bet);
			cknn_bnd2 = (cknn_bet*cknn_res + log(abs(cknn_r / ((double)8.e0*an)))) / (pi*an);
			//ifun=1
			//cv = dcmplx(cknn_s/dl,0.d0)*(cknn_cbnd1+DCMPLX(cknn_bnd2,0.d0))            !1 segment n-tej f.bazowej
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
			cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
			//ifun=3
			//cv = cknn_cbnd1+dcmplx(cknn_bnd2,0.d0)                      ! pot. scalarny
			cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
			//csv1b = .32607257743127307d0*( cv + csv1b ) 
			csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
			// ---- cknn ---- 
			//cv2v = dcqg_b*( cvv1a+cvv1b )
			//cv2s = dcqg_b*( csv1a+csv1b  
			hipDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
			hipDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
			// --------------- cv2 ------------------

			//res_cv = 0.5d0*dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	! dla ifun=1 i ifun=2
			//res_cs = dl*( 1.d0+DLOG(16.d0*an/dl) )/(pi*an)	        ! dla ifun=3
			double res_cs = dl*((double)1.0e0 + log((double)16.0e0*an / dl)) / (pi*an);
			double res_cv = (double)0.5e0*res_cs;
			//cvval2 = cv1v + cv2v + DCMPLX(res_cv,0.d0)     
			//csval2 = cv1s + cv2s + DCMPLX(res_cs,0.d0)
			cvval2 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
			cvval2 = Z_ADD(cv1v, cvval2);
			csval2 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
			csval2 = Z_ADD(cv1s, csval2);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//cvval2 = dcmplx(0.0d0,0.0d0)
			//csval2 = dcmplx(0.0d0,0.0d0)
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		// ------------ Zamiast prcodeury cwire1 -------------------

		//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2 
		//!cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2
		double tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
		//csval2 = csval2/dl
		csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));

		//cvval = cvval = sgnenv*dcmplx(tvecl2,0.d0)*cvval2
		hipDoubleComplex cvval = Z_MUL(cvval2, Z_MAKE((double)sgnenv*tvecl2, (double)0.0e0));
		//csval = -sgnenv*csval2
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)-1.0e0*(double)sgnenv, (double)0.0e0), csval2);

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);
		//hipDoubleComplex cz_ij1 = cz[ i+j*n ];

		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.z,lmc.z);
		//cz[ i+j*n ] = cvval;
		//cz[ i+j*n ] = Z_MAKE(tvecl2,(double)0.0e0);
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] = cz_ij;
		//cz[ i+j*n ] = Z_ADD( cz[ i+j*n ],Z_MAKE((double)10.0f,(double)0.0f) );
		//cz[ i+j*n ] = Z_ADD(cz_ij1, cz_ij );
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_ckmn_re1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1, cvval2, csval2;
	double tvecl1, tvecl2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=1
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 = dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1
			cvval1 = Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                        !ifun=3
			csval1 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 =  cvval1 + dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1	
			//csval1 = csval1 + c_jkr							!ifun=3	
			cvval1 = Z_ADD(Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr), cvval1);
			csval1 = Z_ADD(csval1, c_jkr);

			cvval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval1);
			csval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval1);

			//tvecl1= dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
			tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval1 = csval1/dl
			csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire1 -------------------


		// ------------ Zamiast prcodeury cwire2 -------------------
		ns = ips[k + kk + j + nsmax];
		nlr = ise[ns - 1 + nsmax];
		an = rad[ns - 1];
		double3 rnr = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		rn = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//dlh = (double)0.5e0*dl;
		dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		ise_ms1 = ise[ms - 1];
		ise_ms2 = ise[ms - 1 + nsmax];
		ise_ns1 = ise[ns - 1];
		ise_ns2 = ise[ns - 1 + nsmax];

		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=2
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 = dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2
			cvval2 = Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                            !ifun=3
			csval2 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 =  cvval2 + dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2	
			//csval2 = csval2 + c_jkr								!ifun=3		
			cvval2 = Z_ADD(Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr), cvval2);
			csval2 = Z_ADD(csval2, c_jkr);

			cvval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval2);
			csval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval2);

			//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2
			tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval2 = csval2/dl
			csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2		  
		//csval = csval1 - csval2					!O(rm+) dla k=1, !O(rm-) dla k=2
		hipDoubleComplex cvval = Z_MUL(Z_MAKE(tvecl1, (double)0.0e0), cvval1);
		cvval = Z_ADD(Z_MUL(Z_MAKE(tvecl2, (double)0.0e0), cvval2), cvval);
		hipDoubleComplex csval = Z_SUB(csval1, csval2);


		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);



		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		//cz[ i+j*n ] = cz_ij;
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_ckmn_re2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1, cvval2, csval2;
	double tvecl1, tvecl2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];


		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=1
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 = dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1
			cvval1 = Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                        !ifun=3
			csval1 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 =  cvval1 + dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1	
			//csval1 = csval1 + c_jkr							!ifun=3	
			cvval1 = Z_ADD(Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr), cvval1);
			csval1 = Z_ADD(csval1, c_jkr);

			cvval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval1);
			csval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval1);

			//tvecl1= dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
			tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval1 = csval1/dl
			csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire1 -------------------


		// ------------ Zamiast prcodeury cwire2 -------------------
		ns = ips[k + kk + j + nsmax];
		nlr = ise[ns - 1 + nsmax];
		an = rad[ns - 1];
		double3 rnr = make_double3((double)x[nlr - 1], (double)y[nlr - 1], (double)z[nlr - 1]);
		rn = make_double3((double)x[np - 1], (double)y[np - 1], (double)z[np - 1]);
		rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//dlh = (double)0.5e0*dl;
		dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		ise_ms1 = ise[ms - 1];
		ise_ms2 = ise[ms - 1 + nsmax];
		ise_ns1 = ise[ns - 1];
		ise_ns2 = ise[ns - 1 + nsmax];

		if (war_gpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=2
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 = dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2
			cvval2 = Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                            !ifun=3
			csval2 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 =  cvval2 + dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2	
			//csval2 = csval2 + c_jkr								!ifun=3		
			cvval2 = Z_ADD(Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr), cvval2);
			csval2 = Z_ADD(csval2, c_jkr);

			cvval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval2);
			csval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval2);

			//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2
			tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval2 = csval2/dl
			csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//cvval = dcmplx(tvecl1,0.d0)*cvval1+dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2		  
		//csval = csval1 - csval2					!O(rm+) dla k=1, !O(rm-) dla k=2
		hipDoubleComplex cvval = Z_MUL(Z_MAKE(tvecl1, (double)0.0e0), cvval1);
		cvval = Z_ADD(Z_MUL(Z_MAKE(tvecl2, (double)0.0e0), cvval2), cvval);
		hipDoubleComplex csval = Z_SUB(csval1, csval2);


		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);



		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		//cz[ i+j*n ] = cz_ij;
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_ckmn_im1(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1, cvval2, csval2;
	double tvecl1, tvecl2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i];
		int mlr = ise[ms - 1];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = -1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=1
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 = dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1
			cvval1 = Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                        !ifun=3
			csval1 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 =  cvval1 + dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1	
			//csval1 = csval1 + c_jkr							!ifun=3	
			cvval1 = Z_ADD(Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr), cvval1);
			csval1 = Z_ADD(csval1, c_jkr);

			cvval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval1);
			csval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval1);

			//tvecl1= dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
			tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval1 = csval1/dl
			csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire1 -------------------


		// ------------ Zamiast prcodeury cwire2 -------------------
		ns = ips[k + kk + j + nsmax];
		nlr = ise[ns - 1 + nsmax];
		an = rad[ns - 1];
		double3 rnr = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		rn = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//dlh = (double)0.5e0*dl;
		dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		ise_ms1 = ise[ms - 1];
		ise_ms2 = ise[ms - 1 + nsmax];
		ise_ns1 = ise[ns - 1];
		ise_ns2 = ise[ns - 1 + nsmax];
		//z_ise_ms1 = z[ise[ms-1]];
		//z_ise_ms2 = z[ise[ms-1+nsmax]];
		z_ise_ms1 = z[ise[ms - 1] - 1];
		z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=2
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 = dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2
			cvval2 = Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                            !ifun=3
			csval2 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 =  cvval2 + dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2	
			//csval2 = csval2 + c_jkr								!ifun=3		
			cvval2 = Z_ADD(Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr), cvval2);
			csval2 = Z_ADD(csval2, c_jkr);

			cvval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval2);
			csval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval2);

			//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2
			tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval2 = csval2/dl
			csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//cvval = sgnenv*dcmplx(tvecl1,0.d0)*cvval1 + sgnenv*dcmplx(tvecl2,0.d0)*cvval2   !A(rm+)	dla k=1, A(rm-) dla k=2		  
		//csval = csval1*sgnenv - sgnenv*csval2                      					!O(rm+) dla k=1, !O(rm-) dla k=2
		//
		//hipDoubleComplex cvval = Z_MUL( Z_MAKE((double)sgnenv*tvecl1,(double)0.0e0),cvval1 );
		//cvval = Z_ADD( Z_MUL(Z_MAKE((double)sgnenv*tvecl2,(double)0.0e0),cvval2),cvval );
		hipDoubleComplex cvval = Z_MUL(Z_MAKE(tvecl1, (double)0.0e0), cvval1);
		cvval = Z_ADD(Z_MUL(Z_MAKE(tvecl2, (double)0.0e0), cvval2), cvval);
		cvval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), cvval);
		//hipDoubleComplex csval = Z_MUL( Z_MAKE((double)sgnenv,(double)0.0e0),csval2 );
		//csval = Z_SUB( Z_MUL( Z_MAKE((double)sgnenv,(double)0.0e0),csval1 ),csval );
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), Z_SUB(csval1, csval2));

		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);



		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE((double)sgnenv,tvecl2);
		//cz[ i+j*n ] = cvval;
		//cz[ i+j*n ] = cz_ij;
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//----------------------------------------------------------------------

__global__ void wim_gpu_ckmn_im2(hipDoubleComplex *cz, float *x, float *y, float *z,
	float *rad, int *ipc, int *ips, int *ise, int nsmax,
	int n, double ka, double eta0,
	float sgnx, float sgny, float sgnz, float sgnenv,
	int realSubSliceSize, int k, int kk)
{

	double pi = acos(double(-1.e0));
	hipDoubleComplex cvval1, csval1, cvval2, csval2;
	double tvecl1, tvecl2;

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Accumulate row i of A and column j of B
	int i = tx + bx * blockDim.x;
	int j = ty + by * blockDim.y;

	//c16vec c1 = make_c16vec(Z_ONE,Z_MAKE(0.0f, 1.0f),Z_MUL(Z_MAKE(2.0f, 0.0f),Z_ONE));

	if (i < n  && j < realSubSliceSize) {
		int mp = abs(ipc[i]);
		int ms = ips[i + nsmax];
		int mlr = ise[ms - 1 + nsmax];
		//double am = rad[ms-1];
		double3 rmp = make_double3((double)x[mp - 1], (double)y[mp - 1], (double)z[mp - 1]);
		double3 rmlr = make_double3((double)x[mlr - 1], (double)y[mlr - 1], (double)z[mlr - 1]);
		double3 rmc = make_double3((double)0.5e0*(rmlr.x + rmp.x), (double)0.5e0*(rmlr.y + rmp.y), (double)0.5e0*(rmlr.z + rmp.z));
		float znak = 1.0f;
		double3 lmc = make_double3((double)znak*(rmc.x - rmp.x), (double)znak*(rmc.y - rmp.y), (double)znak*(rmc.z - rmp.z));

		// ------------ Zamiast prcodeury cwire1 -------------------
		//ifun=1 i ifun=2=> res_cv
		//fun=3 => res_cs		  
		//call cwire1_gpu(ms,n,cvval1)             ! obliczanie calki na Sn+  pot.wek.
		//call cwire1(ms,n,csval1)

		//int np = abs(ipc[k-1+kk-1+j]);
		//int ns = ips[k-1+kk-1+j];
		int np = abs(ipc[k + kk + j]);
		int ns = ips[k + kk + j];
		double3 rnp = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		int nlr = ise[ns - 1];
		double an = rad[ns - 1];
		double3 rn = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		double3 rd = make_double3(rnp.x - rn.x, rnp.y - rn.y, rnp.z - rn.z);
		double dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//double dlh = (double)0.5e0*dl;
		double3 dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		int ise_ms1 = ise[ms - 1];
		int ise_ms2 = ise[ms - 1 + nsmax];
		int ise_ns1 = ise[ns - 1];
		int ise_ns2 = ise[ns - 1 + nsmax];
		//float z_ise_ms1 = z[ise[ms-1]];
		//float z_ise_ms2 = z[ise[ms-1+nsmax]];
		float z_ise_ms1 = z[ise[ms - 1] - 1];
		float z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval1 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=1
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 = dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1
			cvval1 = Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                        !ifun=3
			csval1 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval1 =  cvval1 + dcmplx(ckmn_s/dl,0.d0)*c_jkr	!ifun=1	
			//csval1 = csval1 + c_jkr							!ifun=3	
			cvval1 = Z_ADD(Z_MUL(Z_MAKE(ckmn_s / dl, (double)0.0e0), c_jkr), cvval1);
			csval1 = Z_ADD(csval1, c_jkr);

			cvval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval1);
			csval1 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval1);

			//tvecl1= dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z       ! obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
			tvecl1 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval1 = csval1/dl
			csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire1 -------------------


		// ------------ Zamiast prcodeury cwire2 -------------------
		ns = ips[k + kk + j + nsmax];
		nlr = ise[ns - 1 + nsmax];
		an = rad[ns - 1];
		double3 rnr = make_double3((double)sgnx*(double)x[nlr - 1], (double)sgny*(double)y[nlr - 1], (double)sgnz*(double)z[nlr - 1]);
		rn = make_double3((double)sgnx*(double)x[np - 1], (double)sgny*(double)y[np - 1], (double)sgnz*(double)z[np - 1]);
		rd = make_double3(rnr.x - rn.x, rnr.y - rn.y, rnr.z - rn.z);
		dl = sqrt(rd.x*rd.x + rd.y*rd.y + rd.z*rd.z);
		//dlh = (double)0.5e0*dl;
		dc = make_double3(rd.x / dl, rd.y / dl, rd.z / dl);

		ise_ms1 = ise[ms - 1];
		ise_ms2 = ise[ms - 1 + nsmax];
		ise_ns1 = ise[ns - 1];
		ise_ns2 = ise[ns - 1 + nsmax];
		//z_ise_ms1 = z[ise[ms-1]];
		//z_ise_ms2 = z[ise[ms-1+nsmax]];
		z_ise_ms1 = z[ise[ms - 1] - 1];
		z_ise_ms2 = z[ise[ms - 1 + nsmax] - 1];

		if (war_gpu_pec_im(ise_ms1, ise_ms2, ise_ns1, ise_ns2, z_ise_ms1, z_ise_ms2)) {
			//call dcqg(0.d0,dlh,cknn,cv1,4)
			//nord=4		    
			// --------------- cv1 ------------------
			cvval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
			csval2 = Z_MAKE((double)0.0e0, (double)0.0e0);
		}
		else {
			//call dcqg(0.d0,dl,ckmn,cv,2)
			//nord=2
			double dcqg_a = (double)0.5e0*dl;
			double dcqg_b = dl;
			double dcqg_c = (double)0.288675134594812882e0 * dcqg_b;
			//ifun=2
			//y=b*0.5d0*(f(a+c)+f(a-c))
			//dla s=a+c
			double ckmn_s = dcqg_a + dcqg_c;
			double3 ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			double ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//stosujemy wz�r Eulera exp(-ja)=cos(a)-jsin(a) => a=ka*r 
			//c_jkr = cdexp(-cj*ka*ckmn_r)/ckmn_r
			hipDoubleComplex c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 = dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2
			cvval2 = Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr);
			//csval1 = c_jkr	                            !ifun=3
			csval2 = c_jkr;
			//dla s=a-c
			ckmn_s = dcqg_a - dcqg_c;
			ckmn_dr = make_double3(rmc.x - rn.x - ckmn_s*dc.x, rmc.y - rn.y - ckmn_s*dc.y, rmc.z - rn.z - ckmn_s*dc.z);
			ckmn_r = sqrt(an*an + ckmn_dr.x*ckmn_dr.x + ckmn_dr.y*ckmn_dr.y + ckmn_dr.z*ckmn_dr.z);
			//c_jkr = exp(-cj*ka*ckmn_r)/ckmn_r
			//c_jkr = dcmplx( cos(ka*ckmn_r), -sin(ka*ckmn_r) )
			//c_jkr = c_jkr/ckmn_r
			c_jkr = Z_MAKE(cos(ka*ckmn_r) / ckmn_r, -sin(ka*ckmn_r) / ckmn_r);
			//cvval2 =  cvval2 + dcmplx((dl-ckmn_s)/dl,0.d0)*c_jkr	!ifun=2	
			//csval2 = csval2 + c_jkr								!ifun=3		
			cvval2 = Z_ADD(Z_MUL(Z_MAKE((dl - ckmn_s) / dl, (double)0.0e0), c_jkr), cvval2);
			csval2 = Z_ADD(csval2, c_jkr);

			cvval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), cvval2);
			csval2 = Z_MUL(Z_MAKE(dcqg_b*(double)0.5e0, (double)0.0e0), csval2);

			//tvecl2 = dc%x*lmc%x+dc%y*lmc%y+dc%z*lmc%z      ! obliczane !lm+ * 1n- dla k=1,	obliczane  !lm- * 1n- dla k=2
			tvecl2 = dc.x*lmc.x + dc.y*lmc.y + dc.z*lmc.z;
			//csval2 = csval2/dl
			csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));
		}
		// ------------ Zamiast prcodeury cwire2 -------------------

		//cvval = sgnenv*dcmplx(tvecl1,0.d0)*cvval1 + sgnenv*dcmplx(tvecl2,0.d0)*cvval2    !A(rm+)	dla k=1, A(rm-) dla k=2		  
		//csval = csval1*sgnenv - sgnenv*csval2											!O(rm+) dla k=1, !O(rm-) dla k=2
		hipDoubleComplex cvval = Z_MUL(Z_MAKE(tvecl1, (double)0.0e0), cvval1);
		cvval = Z_ADD(Z_MUL(Z_MAKE(tvecl2, (double)0.0e0), cvval2), cvval);
		cvval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), cvval);
		hipDoubleComplex csval = Z_MUL(Z_MAKE((double)sgnenv, (double)0.0e0), Z_SUB(csval1, csval2));


		//cz_ij = cz_ij+ka*ka*cvval + znak * csval
		hipDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
		cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka*ka, (double)0.0e0), cvval), cz_ij);

		//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
		cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0*pi*ka), (double)0.0e0));
		cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);



		//cz_d[i_m+j_n*n] = Z_MAKE(10.0f,0.0f);
		//cz_d[i_m+j_n*n] = Z_MUL ( cz_d[i_m+j_n*n] , (Z_MAKE(10.0f,0.0f));
		//cz_d[i_m+j_n*n] = Z_MAKE((double)i_m,(double)j_n);
		//cz_d[i_m+j_n*n]=make_hipDoubleComplex( (i_m),(j_n) );
		//cz_d[ i+j*n ] =  Z_MAKE(float(i*n+j),0.0f) ;
		//cz[ i+j*n ] = Z_MAKE(x[mp-1],z[mp-1]);
		//cz[ i+j*n ] = Z_MAKE(dc.x,dc.z);
		//cz[ i+j*n ] = cknn_cbnd1;
		//cz[ i+j*n ] = Z_MAKE(cknn_res,cknn_bnd2);
		//cz[ i+j*n ] = cz_ij;
		cz[i + (j + kk)*n] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
		//cz[ i+j*n ] =  Z_ADD(c1.z,Z_MAKE(float(i*n+j),0.0f)) ;
	}
}

//-------------------------------------------------------------------------------

static inline int iabs(int x)
{
	if (x >= 0) {
		return x;
	}
	else {
		return -x;
	}
}

//-------------------------------------------------------------------------------

static inline double delick_cpu(double bet) {

	double a0 = (double)1.38629436112e0;
	double a1 = (double)0.09666344259e0;
	double a2 = (double)0.03590092383e0;
	double a3 = (double)0.03742563713e0;
	double a4 = (double)0.01451196212e0;
	double b0 = (double)0.5e0;
	double b1 = (double)0.12498593597e0;
	double b2 = (double)0.06880248576e0;
	double b3 = (double)0.03328355346e0;
	double b4 = (double)0.00441787012e0;
	double res, a, b;
	double am1 = (double)1.0e0 - bet * bet;
	double am12, am13, am14;

	a = a0 + a1 * am1;
	b = b0 + b1 * am1;

	if (am1 >= (double)1.0e-18) {
		am12 = am1 * am1;
		a = a + a2 * am12;
		b = b + b2 * am12;
		if (am1 >= (double)1.0e-12) {
			am13 = am12 * am1;
			a = a + a3 * am13;
			b = b + b3 * am13;
			if (am1 >= (double)1.0e-9) {
				am14 = am13 * am1;
				a = a + a4 * am14;
				b = b + b4 * am14;
			}
		}
	}

	res = a - b * log(am1);
	return res;
}

//-------------------------------------------------------------------------------

static inline bool war_cpu_pec_re(int ise_ms1, int ise_ms2, int ise_ns1, int ise_ns2)
{
	if ((ise_ns1 == ise_ms1 && ise_ns2 == ise_ms2) || (ise_ns1 == ise_ms2 && ise_ns2 == ise_ms1))
	{
		return true;
	}
	else {
		return false;
	}
}

//-------------------------------------------------------------------------------

static inline bool war_cpu_pec_im(int ise_ms1, int ise_ms2, int ise_ns1, int ise_ns2, float z_ise_ms1, float z_ise_ms2)
{
	if (((ise_ns1 == ise_ms1 && ise_ns2 == ise_ms2) && (z_ise_ms1 == 0.0f && z_ise_ms2 == 0.0f))
		|| ((ise_ns1 == ise_ms2 && ise_ns2 == ise_ms1) && (z_ise_ms1 == 0.0f && z_ise_ms2 == 0.0f)))
	{
		return true;
	}
	else {
		return false;
	}
}

//-------------------------------------------------------------------------------

void cpu_cknn_cwire1_re(clDoubleComplex* cz, float* x, float* y, float* z, float* rad,
	int* ipc, int* ips, int* ise, int nsmax, int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = 3.14159265358979323846e0;
	clDoubleComplex cvval1, csval1;
	//double tvecl1;


	for (int k1 = 0; k1 < 2; k1++)
	{
		for (int i = 0; i < n; i++)
		{
			int mp = iabs(ipc[i]);
			//int ms = ips[i];
			int ms = ips[i + k1 * nsmax];
			int mlr = ise[ms - 1 + k1 * nsmax];
			//int mlr = ise[ms-1];
			double rmp_x = (double)x[mp - 1];
			double rmp_y = (double)y[mp - 1];
			double rmp_z = (double)z[mp - 1];
			double rmlr_x = (double)x[mlr - 1];
			double rmlr_y = (double)y[mlr - 1];
			double rmlr_z = (double)z[mlr - 1];
			double rmc_x = (double)0.5e0 * (rmlr_x + rmp_x);
			double rmc_y = (double)0.5e0 * (rmlr_y + rmp_y);
			double rmc_z = (double)0.5e0 * (rmlr_z + rmp_z);
			//double_3 rmp = make_double_3( (double)x[mp-1], (double)y[mp-1], (double)z[mp-1] );
			//double_3 rmlr = make_double_3( (double)x[mlr-1], (double)y[mlr-1], (double)z[mlr-1] );
			//double_3 rmc = make_double_3( (double)0.5e0*(rmlr.x+rmp.x), (double)0.5e0*(rmlr.y+rmp.y), (double)0.5e0*(rmlr.z+rmp.z) );
			//float znak = -1.0f;
			float znak = 2.0f * (float)k1 - 1.0f;
			double lmc_x = (double)znak * (rmc_x - rmp_x);
			double lmc_y = (double)znak * (rmc_y - rmp_y);
			double lmc_z = (double)znak * (rmc_z - rmp_z);
			//double_3 lmc = make_double_3( (double)znak*(rmc.x-rmp.x), (double)znak*(rmc.y-rmp.y), (double)znak*(rmc.z-rmp.z) );

			for (int j = 0; j < realSubSliceSize; j++)
			{
				// ------------ Zamiast prcodeury cwire1 -------------------
				// Obliczanie calki na Sn+  pot.wek.
				int np = iabs(ipc[k + kk + j]);
				int ns = ips[k + kk + j];
				//int np = iabs(ipc[j]);
				//int ns = ips[j];
				double rnp_x = (double)x[np - 1];
				double rnp_y = (double)y[np - 1];
				double rnp_z = (double)z[np - 1];
				//double_3 rnp = make_double_3( (double)x[np-1],(double)y[np-1],(double)z[np-1] );
				int nlr = ise[ns - 1];
				double an = rad[ns - 1];
				double rn_x = (double)x[nlr - 1];
				double rn_y = (double)y[nlr - 1];
				double rn_z = (double)z[nlr - 1];
				double rd_x = rnp_x - rn_x;
				double rd_y = rnp_y - rn_y;
				double rd_z = rnp_z - rn_z;
				//double_3 rn = make_double_3( (double)x[nlr-1],(double)y[nlr-1],(double)z[nlr-1] );
				//double_3 rd = make_double_3( rnp.x-rn.x, rnp.y-rn.y, rnp.z-rn.z );
				double dl = sqrt(rd_x * rd_x + rd_y * rd_y + rd_z * rd_z);
				double dlh = (double)0.5e0 * dl;
				double dc_x = rd_x / dl;
				double dc_y = rd_y / dl;
				double dc_z = rd_z / dl;
				//double_3 dc = make_double_3( rd.x/dl, rd.y/dl, rd.z/dl );

				int ise_ms1 = ise[ms - 1];
				int ise_ms2 = ise[ms - 1 + nsmax];
				int ise_ns1 = ise[ns - 1];
				int ise_ns2 = ise[ns - 1 + nsmax];

				if (war_cpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
					// --------------- cv1 ------------------
					double dcqg_a = (double)0.5e0 * dlh;
					double dcqg_b = dlh;
					double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
					//s=a+c
					double cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					double cknn_r = (double)0.5e0 * dl - cknn_s;
					double cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					clDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					double cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					double cknn_res = delick_cpu(cknn_bet);
					double cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					clDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					clDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					clDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));  // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
					// ---- cknn ----
					dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					clDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					clDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // !1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
					// ---- cknn ----
					clDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
					clDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
					// --------------- cv1 ------------------

					// --------------- cv2 ------------------
					dcqg_a = (double)0.5e0 * (dlh + dl);
					dcqg_b = dl - dlh;
					dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
					// ---- cknn ----
					dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
					// ---- cknn ----
					clDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
					clDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
					// --------------- cv2 ------------------

					double res_cs = dl * ((double)1.0e0 + log((double)16.0e0 * an / dl)) / (pi * an);
					double res_cv = (double)0.5e0 * res_cs;
					cvval1 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
					cvval1 = Z_ADD(cv1v, cvval1);
					csval1 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
					csval1 = Z_ADD(cv1s, csval1);
				}
				else {
					cvval1 = Z_ZERO;
					csval1 = Z_ZERO;
				}
				// ------------ Zamiast prcodeury cwire1 -------------------

				double tvecl1 = dc_x * lmc_x + dc_y * lmc_y + dc_z * lmc_z;   // obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=1
				csval1 = Z_DIV(csval1, Z_MAKE(dl, (double)0.0e0));
				clDoubleComplex cvval = Z_MUL(cvval1, Z_MAKE(tvecl1, (double)0.0e0));
				clDoubleComplex csval = csval1;
				//cz_ij = cz_ij+ka*ka*cvval + znak * csval
				clDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
				cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka * ka, (double)0.0e0), cvval), cz_ij);
				//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
				cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0 * pi * ka), (double)0.0e0));
				cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);

				//cz[ i+(j+kk)*n ] = Z_ADD(cz[i + (j + kk)*n], cz_ij);


				//clDoubleComplex cz_ij = Z_MAKE((double)0.0e0,(double)0.0e0);  //do testow
				//clDoubleComplex cz_ij	= cvval;
				//clDoubleComplex cz_ij = Z_MAKE((double)i,(double)j);
				//clDoubleComplex cz_ij = Z_MAKE(tvecl1,tvecl2);
				//clDoubleComplex cz_ij = Z_MAKE(rmp_z,rmlr_z);
				//cz[i + (j + kk)*n] = Z_MAKE(rmc_z,lmc_z);
				//cz[i + (j + kk)*n] = Z_MAKE(tvecl1,tvecl2);
				//cz[i + (j + kk)*n] = Z_MAKE((double)i,(double)j);
				//clDoubleComplex cz_ij = Z_MAKE((double)i,(double)j);
				cz[i + (j + kk) * n] = Z_ADD(cz[i + (j + kk) * n], cz_ij);
			}
		}
	}


	//double sin_v = sin(pi/4);
	//printf("Got sin value: %.14lf \n", sin_v);

}

//-------------------------------------------------------------------------------

void cpu_cknn_cwire2_re(clDoubleComplex* cz, float* x, float* y, float* z, float* rad,
	int* ipc, int* ips, int* ise, int nsmax, int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

	double pi = 3.14159265358979323846e0;
	clDoubleComplex cvval2, csval2;
	//double tvecl1;


	for (int k1 = 0; k1 < 2; k1++)
	{
		for (int i = 0; i < n; i++)
		{
			int mp = iabs(ipc[i]);
			//int ms = ips[i];
			int ms = ips[i + k1 * nsmax];
			int mlr = ise[ms - 1 + k1 * nsmax];
			//int mlr = ise[ms-1];
			double rmp_x = (double)x[mp - 1];
			double rmp_y = (double)y[mp - 1];
			double rmp_z = (double)z[mp - 1];
			double rmlr_x = (double)x[mlr - 1];
			double rmlr_y = (double)y[mlr - 1];
			double rmlr_z = (double)z[mlr - 1];
			double rmc_x = (double)0.5e0 * (rmlr_x + rmp_x);
			double rmc_y = (double)0.5e0 * (rmlr_y + rmp_y);
			double rmc_z = (double)0.5e0 * (rmlr_z + rmp_z);
			//double_3 rmp = make_double_3( (double)x[mp-1], (double)y[mp-1], (double)z[mp-1] );
			//double_3 rmlr = make_double_3( (double)x[mlr-1], (double)y[mlr-1], (double)z[mlr-1] );
			//double_3 rmc = make_double_3( (double)0.5e0*(rmlr.x+rmp.x), (double)0.5e0*(rmlr.y+rmp.y), (double)0.5e0*(rmlr.z+rmp.z) );
			//float znak = -1.0f;
			float znak = 2.0f * (float)k1 - 1.0f;
			double lmc_x = (double)znak * (rmc_x - rmp_x);
			double lmc_y = (double)znak * (rmc_y - rmp_y);
			double lmc_z = (double)znak * (rmc_z - rmp_z);
			//double_3 lmc = make_double_3( (double)znak*(rmc.x-rmp.x), (double)znak*(rmc.y-rmp.y), (double)znak*(rmc.z-rmp.z) );

			for (int j = 0; j < realSubSliceSize; j++)
			{
				// ------------ Zamiast prcodeury cwire2 -------------------
				// Obliczanie calki na Sn+  pot.wek.
				int np = iabs(ipc[k + kk + j]);
				int ns = ips[k + kk + j + nsmax];
				//int np = iabs(ipc[j]);
				//int ns = ips[j];
				int nlr = ise[ns - 1 + nsmax];
				double an = rad[ns - 1];
				//double_3 rn = make_double_3( (double)x[np-1],(double)y[np-1],(double)z[np-1] );
				double rn_x = (double)x[np - 1];
				double rn_y = (double)y[np - 1];
				double rn_z = (double)z[np - 1];
				double rnr_x = (double)x[nlr - 1];
				double rnr_y = (double)y[nlr - 1];
				double rnr_z = (double)z[nlr - 1];
				double rd_x = rnr_x - rn_x;
				double rd_y = rnr_y - rn_y;
				double rd_z = rnr_z - rn_z;
				//double_3 rn = make_double_3( (double)x[nlr-1],(double)y[nlr-1],(double)z[nlr-1] );
				//double_3 rd = make_double_3( rnp.x-rn.x, rnp.y-rn.y, rnp.z-rn.z );
				double dl = sqrt(rd_x * rd_x + rd_y * rd_y + rd_z * rd_z);
				double dlh = (double)0.5e0 * dl;
				double dc_x = rd_x / dl;
				double dc_y = rd_y / dl;
				double dc_z = rd_z / dl;
				//double_3 dc = make_double_3( rd.x/dl, rd.y/dl, rd.z/dl );

				int ise_ms1 = ise[ms - 1];
				int ise_ms2 = ise[ms - 1 + nsmax];
				int ise_ns1 = ise[ns - 1];
				int ise_ns2 = ise[ns - 1 + nsmax];

				if (war_cpu_pec_re(ise_ms1, ise_ms2, ise_ns1, ise_ns2)) {
					// --------------- cv1 ------------------
					double dcqg_a = (double)0.5e0 * dlh;
					double dcqg_b = dlh;
					double dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
					//s=a+c
					double cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					double cknn_r = (double)0.5e0 * dl - cknn_s;
					double cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					clDoubleComplex cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					double cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					double cknn_res = delick_cpu(cknn_bet);
					double cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					clDoubleComplex cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					clDoubleComplex cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					clDoubleComplex csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));  // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
					// ---- cknn ----
					dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					clDoubleComplex cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					clDoubleComplex csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // !1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
					// ---- cknn ----
					clDoubleComplex cv1v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
					clDoubleComplex cv1s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
					// --------------- cv1 ------------------

					// --------------- cv2 ------------------
					dcqg_a = (double)0.5e0 * (dlh + dl);
					dcqg_b = dl - dlh;
					dcqg_c = (double)0.43056815579702629e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cvv1a = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					csv1a = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, cvv1a));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1a = Z_MUL(Z_MAKE((double)0.17392742256872693e0, (double)0.0e0), Z_ADD(cv, csv1a));
					// ---- cknn ----
					dcqg_c = (double)0.16999052179242813e0 * dcqg_b;
					//s=a+c
					cknn_s = dcqg_a + dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cvv1b = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					//ifun=3
					csv1b = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					// ---- cknn ----
					//s=a-c
					cknn_s = dcqg_a - dcqg_c;
					// ---- cknn ----
					cknn_r = (double)0.5e0 * dl - cknn_s;
					cknn_rr = sqrt(an * an + cknn_r * cknn_r);
					cknn_cbnd1 = Z_MAKE((cos(ka * cknn_rr) - (double)1.0e0) / cknn_rr, -sin(ka * cknn_rr) / cknn_rr);
					cknn_bet = (an + an) / sqrt((double)4.0e0 * an * an + cknn_r * cknn_r);
					cknn_res = delick_cpu(cknn_bet);
					cknn_bnd2 = (cknn_bet * cknn_res + log(fabs(cknn_r / ((double)8.e0 * an)))) / (pi * an);
					//ifun=1
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // 1 segment n-tej f.bazowej
					cv = Z_MUL(Z_MAKE(cknn_s / dl, (double)0.0e0), cv);
					cvv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, cvv1b));
					//ifun=3
					cv = Z_ADD(cknn_cbnd1, Z_MAKE(cknn_bnd2, (double)0.0e0));   // pot. scalarny
					csv1b = Z_MUL(Z_MAKE((double)0.32607257743127307e0, (double)0.0e0), Z_ADD(cv, csv1b));
					// ---- cknn ----
					clDoubleComplex cv2v = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(cvv1a, cvv1b));
					clDoubleComplex cv2s = Z_MUL(Z_MAKE(dcqg_b, (double)0.0e0), Z_ADD(csv1a, csv1b));
					// --------------- cv2 ------------------

					double res_cs = dl * ((double)1.0e0 + log((double)16.0e0 * an / dl)) / (pi * an);
					double res_cv = (double)0.5e0 * res_cs;
					cvval2 = Z_ADD(cv2v, Z_MAKE(res_cv, (double)0.0e0));
					cvval2 = Z_ADD(cv1v, cvval2);
					csval2 = Z_ADD(cv2s, Z_MAKE(res_cs, (double)0.0e0));
					csval2 = Z_ADD(cv1s, csval2);
				}
				else {
					cvval2 = Z_ZERO;
					csval2 = Z_ZERO;
				}
				// ------------ Zamiast prcodeury cwire2 -------------------

				double tvecl2 = dc_x * lmc_x + dc_y * lmc_y + dc_z * lmc_z;   // obliczane !lm+ * 1n+ dla k=1, obliczane !lm- * 1n+ dla k=2
				csval2 = Z_DIV(csval2, Z_MAKE(dl, (double)0.0e0));
				clDoubleComplex cvval = Z_MUL(cvval2, Z_MAKE(tvecl2, (double)0.0e0));
				//csval = -csval2
				clDoubleComplex csval = Z_MUL(Z_MAKE((double)-1.0e0, (double)0.0e0), csval2);
				//cz_ij = cz_ij+ka*ka*cvval + znak * csval
				clDoubleComplex cz_ij = Z_MUL(Z_MAKE((double)znak, (double)0.0e0), csval);
				cz_ij = Z_ADD(Z_MUL(Z_MAKE(ka * ka, (double)0.0e0), cvval), cz_ij);
				//cz_ij = cz_ij*cj*eta0/(4.0e0*pi*ka)
				cz_ij = Z_MUL(cz_ij, Z_MAKE(eta0 / ((double)4.0e0 * pi * ka), (double)0.0e0));
				cz_ij = Z_MUL(cz_ij, Z_ONE_IMG);

				//cz[ i+(j+kk)*n ] = Z_ADD(cz[i + (j + kk)*n], cz_ij);
				//clDoubleComplex cz_ij = Z_MAKE((double)0.0e0,(double)0.0e0);  //do testow
				//clDoubleComplex cz_ij	= cvval;
				//clDoubleComplex cz_ij = Z_MAKE((double)i,(double)j);
				//clDoubleComplex cz_ij = Z_MAKE(tvecl1,tvecl2);
				//clDoubleComplex cz_ij = Z_MAKE(rmp_z,rmlr_z);
				//cz[i + (j + kk)*n] = Z_MAKE(rmc_z,lmc_z);
				//cz[i + (j + kk)*n] = Z_MAKE(tvecl1,tvecl2);
				//cz[i + (j + kk)*n] = Z_MAKE((double)i,(double)j);
				//clDoubleComplex cz_ij = Z_MAKE((double)i,(double)j);
				cz[i + (j + kk) * n] = Z_ADD(cz[i + (j + kk) * n], cz_ij);
			}
		}
	}


	//double sin_v = sin(pi/4);
	//printf("Got sin value: %.14lf \n", sin_v);

}

//----------------------------------------------------------------------

void cpu_cknn_cwire1_im(clDoubleComplex* cz, float* x, float* y, float* z, float* rad,
	int* ipc, int* ips, int* ise, int nsmax, int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

}

//----------------------------------------------------------------------

void cpu_cknn_cwire2_im(clDoubleComplex* cz, float* x, float* y, float* z, float* rad,
	int* ipc, int* ips, int* ise, int nsmax, int n, double ka, double eta0,
	int realSubSliceSize, int k, int kk)
{

}

//----------------------------------------------------------------------

void zmatrix_gpu(hipDoubleComplex *cz_h, float *x_h, float *y_h, float *z_h, float *rad_h,
	int *ipc_h, int *ips_h, int *ise_h, int nsmax, double ka, int npls,
	int ienv, float sgnx, float sgny, float sgnz, float sgnenv, int rank)
{

	double pi = acos(-1.0);
	double eta0 = (double)120.*pi;
	hipDoubleComplex cjq = Z_MAKE(0.0, eta0 / (4.0*pi*ka));

	//std::cout << std::endl;
	//std::cout << "  --------------------------------------------------------------------------" << std::endl;
	//std::cout << "  Initializing hardware-accelerated zmatrix assembly phase of wire-grid MoM " << std::endl;
	//std::cout << "  --------------------------------------------------------------------------" << std::endl;

	float *x_d, *y_d, *z_d, *rad_d;
	int *ipc_d, *ips_d, *ise_d;
	hipDoubleComplex *cz_d;
	dim3 threads, grid;
	float elapsedTime_kernel;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEvent_t start_kernel, stop_kernel;

	hipMalloc((void**)&x_d, nsmax * sizeof(float));
	hipMalloc((void**)&y_d, nsmax * sizeof(float));
	hipMalloc((void**)&z_d, nsmax * sizeof(float));
	hipMalloc((void**)&rad_d, nsmax * sizeof(float));
	hipMalloc((void**)&ipc_d, nsmax * sizeof(int));
	hipMalloc((void**)&ips_d, 2 * nsmax * sizeof(int));
	hipMalloc((void**)&ise_d, 2 * nsmax * sizeof(int));
	CUDA_CALL(hipMalloc((void**)&cz_d, npls * npls * sizeof(hipDoubleComplex)), "hipMalloc failed !\n");

	//Open file where results are saved
	FILE *f = fopen("GPU time.dat", "w+");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	//Prepare events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);

	// Start record (data transfer and kernel execution)
	hipEventRecord(start, 0);

	hipMemcpy(x_d, x_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(rad_d, rad_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ipc_d, ipc_h, nsmax * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ips_d, ips_h, 2 * nsmax * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ise_d, ise_h, 2 * nsmax * sizeof(int), hipMemcpyHostToDevice);

	// Setup execution parameters
	threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
	//threads = dim3(2*BLOCK_SIZE,BLOCK_SIZE);
	//grid = dim3( 1+(int)ceil(float(N-1)/float(threads.x)), 1+(int)ceil(float(N-1)/float(threads.y)) );
	grid = dim3(((npls - 1) / threads.x) + 1, ((npls - 1) / threads.y) + 1);

	//printf(" |\n");
	//printf("  --> set grid size %d  %d \n",grid.x,grid.y); 
	//printf("\n");

	// Start record (data transfer and kernel execution)
	//hipEventRecord(start, 0);

	// Start record (kernel execution only)
	hipEventRecord(start_kernel, 0);

	// Execute the kernels //  
	wim_gpu_cknn_cwire1_re1 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);   // device 1 when multi-GPU or device 1 and device 2 with another SubSlice level
	wim_gpu_cknn_cwire1_re2 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);   // device 2 
	// ----- PEC -----
	if (ienv == 2){
		wim_gpu_cknn_cwire1_im1 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
		wim_gpu_cknn_cwire1_im2 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
	}
	// ----- PEC -----
	wim_gpu_cknn_cwire2_re1 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);
	wim_gpu_cknn_cwire2_re2 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);
	if (ienv == 2){
		wim_gpu_cknn_cwire2_im1 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
		wim_gpu_cknn_cwire2_im2 <<< grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
	}
	// ----- PEC -----
	wim_gpu_ckmn_re1 <<<grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);
	wim_gpu_ckmn_re2 <<<grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, npls, 0, 0);
	// ----- PEC -----
	if (ienv == 2){
		wim_gpu_ckmn_im1 <<<grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
		wim_gpu_ckmn_im2 <<<grid, threads >>>(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, npls, 0, 0);
	}
	// ----- PEC -----  

	// Stop event (kernel execution only)
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	hipEventElapsedTime(&elapsedTime_kernel, start_kernel, stop_kernel);

	std::cout << std::endl << " Process - " << rank << " zmatrix GPU execution time = " << std::setprecision(4) << elapsedTime_kernel / 1.0e3 << " s" << std::endl;

	// Copy impedance matrix from GPU to CPU
	hipMemcpy(cz_h, cz_d, sizeof(hipDoubleComplex) * npls*npls, hipMemcpyDeviceToHost);

	// Stop event (data transfer and kernel execution)
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	std::cout << std::endl << " Process - " << rank << " zmatrix GPU execution time including data transfer = " << std::setprecision(4) << elapsedTime / 1.0e3 << " s" << std::endl;
	std::cout << std::endl << " Process - " << rank << " zmatrix GPU data transfer time = " << std::setprecision(4) << (elapsedTime - elapsedTime_kernel) / 1.0e3 << " s" << std::endl;

	fprintf(f, "GPU execution time = %.3f ms\n", elapsedTime_kernel / 1.0e0);
	fprintf(f, "GPU execution time including data transfer = %.3f ms\n", elapsedTime / 1.0e0);
	fprintf(f, "Data transfer time = %.3f ms\n", (elapsedTime - elapsedTime_kernel) / 1.0e0);
	fclose(f);

	//hipDeviceSynchronize();

	//Destroy events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);


	hipFree(cz_d);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
	hipFree(rad_d);
	hipFree(ipc_d);
	hipFree(ips_d);
	hipFree(ise_d);


} 

//----------------------------------------------------------------------

void zmatrix_gpu_ol(hipDoubleComplex *cz_h, float *x_h, float *y_h, float *z_h, float *rad_h,
	int *ipc_h, int *ips_h, int *ise_h, int nsmax, double ka, int npls,
	int ienv, float sgnx, float sgny, float sgnz, float sgnenv, int slices, int rank)
{

	double pi = acos(-1.0);
	double eta0 = (double)120.*pi;
	hipDoubleComplex cjq = Z_MAKE(0.0, eta0 / (4.0*pi*ka));

	float *x_d, *y_d, *z_d, *rad_d;
	int *ipc_d, *ips_d, *ise_d;
	hipDoubleComplex *cz_d;
	dim3 threads, grid;
	float elapsedTime;
	hipEvent_t start, stop;
	int const nStreams = 4;

	//std::cout << std::endl;
	//std::cout << "  --------------------------------------------------------------------------" << std::endl;
	//std::cout << "  Initializing hardware-accelerated zmatrix assembly phase of wire-grid MoM " << std::endl;
	//std::cout << "  --------------------------------------------------------------------------" << std::endl;

	//Open file where results are saved
	FILE *f = fopen("GPU time.dat", "w+");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	// allocate and initialize an array of stream handles
	hipStream_t *streams = (hipStream_t *)malloc(nStreams*sizeof(hipStream_t));

	//Prepare events
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Create streams
	for (int i = 0; i < nStreams; i++)
		hipStreamCreate(&(streams[i]));


	hipMalloc((void**)&x_d, nsmax * sizeof(float));
	hipMalloc((void**)&y_d, nsmax * sizeof(float));
	hipMalloc((void**)&z_d, nsmax * sizeof(float));
	hipMalloc((void**)&rad_d, nsmax * sizeof(float));
	hipMalloc((void**)&ipc_d, nsmax * sizeof(int));
	hipMalloc((void**)&ips_d, 2 * nsmax * sizeof(int));
	hipMalloc((void**)&ise_d, 2 * nsmax * sizeof(int));
	//hipMalloc( (void**)&cz_d, n * n * sizeof(hipDoubleComplex) ); 

	// Start record event
	hipEventRecord(start, 0);

	hipMemcpy(x_d, x_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(z_d, z_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(rad_d, rad_h, nsmax * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ipc_d, ipc_h, nsmax * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ips_d, ips_h, 2 * nsmax * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ise_d, ise_h, 2 * nsmax * sizeof(int), hipMemcpyHostToDevice);


	int SliceSize = (int)ceil((double)npls / slices);

	int SubSlices = nStreams;

	for (int i = 0; i < npls; i += SliceSize) {


		int RealSliceSize = min(npls - i, SliceSize);
		//printf("RealSliceSize:   %d\n", RealSliceSize);

		hipMalloc((void**)&cz_d, npls * RealSliceSize * sizeof(hipDoubleComplex));

		int SubSliceSize = (int)ceil((double)RealSliceSize / SubSlices);
		int ii = 0;

		// Setup execution parameters
		//threads = dim3(BLOCK_SIZE,BLOCK_SIZE);
		threads = dim3(2 * BLOCK_SIZE, BLOCK_SIZE);
		grid = dim3(((npls - 1) / threads.x) + 1, ((SubSliceSize - 1) / threads.y) + 1);


		for (int j = 0; j < nStreams; j++) {

			int RealSubSliceSize = min(RealSliceSize - ii, SubSliceSize);
			//printf("RealSubSliceSize:   %d\n", RealSubSliceSize);

			// Execute the kernels //  
			wim_gpu_cknn_cwire1_re1 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);   // device 1 when multi-GPU or device 1 and device 2 with another SubSlice level
			wim_gpu_cknn_cwire1_re2 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);   // device 2 
			// ----- PEC -----
			if (ienv == 2){
				wim_gpu_cknn_cwire1_im1 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
				wim_gpu_cknn_cwire1_im2 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
			}
			// ----- PEC -----
			wim_gpu_cknn_cwire2_re1 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);
			wim_gpu_cknn_cwire2_re2 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);
			// ----- PEC -----
			if (ienv == 2){
				wim_gpu_cknn_cwire2_im1 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
				wim_gpu_cknn_cwire2_im2 <<< grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
			}
			// ----- PEC -----
			wim_gpu_ckmn_re1 << < grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);
			wim_gpu_ckmn_re2 << < grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, RealSubSliceSize, i, ii);
			// ----- PEC -----
			if (ienv == 2){
				wim_gpu_ckmn_im1 << < grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
				wim_gpu_ckmn_im2 << < grid, threads, 0, streams[j] >> >(cz_d, x_d, y_d, z_d, rad_d, ipc_d, ips_d, ise_d, nsmax, npls, ka, eta0, sgnx, sgny, sgnz, sgnenv, RealSubSliceSize, i, ii);
			}
			// ----- PEC -----

			// Copy impedance matrix from GPU to CPU
			//hipMemcpyAsync( cz_h+((i+ii)*n), cz_d+(ii*n), n*RealSubSliceSize*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, streams[j] );

			ii += SubSliceSize;
		}


		// Copy impedance matrix from GPU to CPU
		//hipMemcpy( cz_h+(i*n), cz_d, sizeof(hipDoubleComplex) * n*RealSliceSize, hipMemcpyDeviceToHost );

		ii = 0;
		//	printf("\n");


		for (int j = 0; j < nStreams; j++) {
			int RealSubSliceSize = min(RealSliceSize - ii, SubSliceSize);
			//	printf("RealSubSliceSize:   %d, (i+ii)*n: %d \n", RealSubSliceSize, (i+ii)*n );
			hipMemcpyAsync(cz_h + ((i + ii)*npls), cz_d + (ii*npls), npls*RealSubSliceSize*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, streams[j]);
			ii += SubSliceSize;
		}


		hipFree(cz_d);

	}

		//Synchronize streams
		for (int i = 0; i < nStreams; i++)
			hipStreamSynchronize(streams[i]);

		// Stop record event
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);

		//printf("\n");
		//printf("  GPU execution time = %.4f s\n",kernel_total/1.0e3);
		printf("\n");
		printf("  GPU execution time including data transfer = %.4f s\n", elapsedTime / 1.0e3);
		printf("\n");
		//printf("  Data transfer time = %.4f s\n",(data_total)/1.0e3);    
		//printf("\n");

		//fprintf(f, "GPU execution time = %.3f ms\n",kernel_total/1.0e0);
		fprintf(f, "GPU execution time including data transfer = %.3f ms\n", elapsedTime / 1.0e0);
		//fprintf(f, "Data transfer time = %.3f ms\n",(data_total)/1.0e0);
		fclose(f);


		//Destroy events
		hipEventDestroy(start);
		hipEventDestroy(stop);

		//Destroy streams
		for (int i = 0; i < nStreams; i++)
			hipStreamDestroy(streams[i]);


		free(streams);
		hipFree(x_d);
		hipFree(y_d);
		hipFree(z_d);
		hipFree(rad_d);
		hipFree(ipc_d);
		hipFree(ips_d);
		hipFree(ise_d);

}

//----------------------------------------------------------------------


/* hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
} */
